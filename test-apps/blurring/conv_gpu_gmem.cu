#include "hip/hip_runtime.h"
#include "conv_gpu.h"

#include <assert.h>

// ===================
//    GLOBAL MEMORY
// ===================
__global__ void conv_h_gpu_gmem_kernel(
        unsigned int* src,
        unsigned int* dst,
        int w,
        int h,
        size_t spitch,
        size_t dpitch,
        int ks,
        float* kernelbuffer) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < w && y < h) {
        float rr = 0.0f, gg = 0.0f, bb = 0.0f;

        for (int i = 0; i < ks; i++) {
            int xx = x + (i - ks / 2);

            // clamp
            xx = max<int>(min<int>(xx, w-1), 0);

            // get pixel
            unsigned int pixel = src[y * spitch + xx];

            // get colors
            unsigned char r = pixel & 0xff;
            unsigned char g = (pixel >> 8) & 0xff;
            unsigned char b = (pixel >> 16) & 0xff;

            rr += r * kernelbuffer[i];
            gg += g * kernelbuffer[i];
            bb += b * kernelbuffer[i];
        }
        unsigned char rr_c = rr + 0.5f;
        unsigned char gg_c = gg + 0.5f;
        unsigned char bb_c = bb + 0.5f;
        dst[y * dpitch + x] = rr_c | (gg_c << 8) | (bb_c << 16);
    }
}

__global__ void conv_v_gpu_gmem_kernel(
        unsigned int* src,
        unsigned int* dst,
        int w,
        int h,
        size_t spitch,
        size_t dpitch,
        int ks,
        float* kernelbuffer) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
        float rr = 0.0f, gg = 0.0f, bb = 0.0f;

        for (int i = 0; i < ks; i++) {
            int yy = y + (i - ks / 2);

            // clamp
            yy = max<int>(min<int>(yy, h-1), 0);

            // get pixel
            unsigned int pixel = src[yy * spitch + x];

            // get colors
            unsigned char r = pixel & 0xff;
            unsigned char g = (pixel >> 8) & 0xff;
            unsigned char b = (pixel >> 16) & 0xff;

            rr += r * kernelbuffer[i];
            gg += g * kernelbuffer[i];
            bb += b * kernelbuffer[i];
        }

        unsigned char rr_c = rr + 0.5f;
        unsigned char gg_c = gg + 0.5f;
        unsigned char bb_c = bb + 0.5f;

        dst[y * dpitch + x] = rr_c | (gg_c << 8) | (bb_c << 16);
    }
}

void conv_h_gpu_gmem(image_gpu &dst, const image_gpu &src, const filterkernel_gpu &kernel) {
    assert(dst.width == src.width);
    assert(dst.height == src.height);

    dim3 dimBlock(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    dim3 dimGrid(div_up(src.width, dimBlock.x), div_up(src.height, dimBlock.y));
    conv_h_gpu_gmem_kernel<<<dimGrid, dimBlock>>>(
            src.data,
            dst.data,
            src.width,
            src.height,
            src.pitch,
            dst.pitch,
            kernel.ks,
            kernel.data);
    CUDA_CHECK_ERROR;
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR;
}

void conv_v_gpu_gmem(image_gpu &dst, const image_gpu &src, const filterkernel_gpu &kernel) {
    dim3 dimBlock(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    dim3 dimGrid(div_up(src.width, dimBlock.x), div_up(src.height, dimBlock.y));
    conv_v_gpu_gmem_kernel<<<dimGrid, dimBlock>>>(
            src.data,
            dst.data,
            src.width,
            src.height,
            src.pitch,
            dst.pitch,
            kernel.ks,
            kernel.data);
    CUDA_CHECK_ERROR;
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR;
}