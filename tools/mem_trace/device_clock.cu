#include "hip/hip_runtime.h"
#include <stdint.h>
#include <nvbit.h>
#include <hip/hip_runtime.h>

#include <memtrack/util.h>


__device__ uint64_t get_global_time()
{
    uint64_t result;

    // sm30 or higher is required for %globaltimer which is a nanosecond realtime clock.
    // see https://nvidia.github.io/libcudacxx/standard_api/time_library/chrono.html
    // and https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#special-registers-globaltimer

    // see [^1] for contraint letters and %-escaping
    // https://docs.nvidia.com/cuda/inline-ptx-assembly/index.html
    asm (
        "mov.u64 %0, %%globaltimer;" : "=l"(result)
    );

    return result;
}

__global__ void probe_device_clock(uint64_t *global_time)
{
    *global_time = get_global_time();
}

uint64_t get_time_difference()
{
    int64_t device_time;


    void *buf = hipMalloc(sizeof(device_time));

    int64_t host_time = util::time_to_ns(util::now());
    probe_device_clock<<<1, 1>>>(buf);

    hipMemcpy(&device_time, buf, sizeof(device_time), hipMemcpyDeviceToHost);

    return device_time - host_time;
}